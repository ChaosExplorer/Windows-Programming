#include "hip/hip_runtime.h"
#include<limits>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include <unistd.h>
#include "cuda_pairhmm.h"

#define BSIZE 128
#define BALIGN(a) (((a)+127)/128*128)

using namespace std;
using namespace cuda_pairhmm;

struct NUM_ADD
{
    short2 read_haplotype_number;
    int address_array;
};

__global__ void  pairHMM( int size, char * data,  NUM_ADD * num_add, float * result,float * MG,float * DG, float * IG)
{
    int offset=blockIdx.x;

    __shared__ short2 read_haplotype_number;
    __shared__ char * read_base_array;
    __shared__ float * parameter_array;
    __shared__ char4 * haplotype_base_array;
    __shared__ float * MGG;
    __shared__ float * IGG;
    __shared__ float * DGG;
    if(threadIdx.x==0)
    {
        MGG=MG+offset*500;
        DGG=IG+offset*500;
        IGG=DG+offset*500;
    }

    while(offset<size)
    {
        float result_block=0;
        __shared__ int round;
        __shared__ int skip;

        //as each time it will deal with 2 read&haplotype pairs
        // each block deal with one pairs of haplotype & read

        if( threadIdx.x==0)
        {
            read_haplotype_number=num_add[offset].read_haplotype_number;
            read_base_array=(char *) (data+num_add[offset].address_array);
            parameter_array=(float *) (read_base_array+(read_haplotype_number.x+BSIZE-1)/BSIZE*BSIZE);
            skip=(sizeof(float)*read_haplotype_number.x+BSIZE-1)/BSIZE*BSIZE/sizeof(float);
            haplotype_base_array=(char4 *) (parameter_array+skip*4);
            round=(read_haplotype_number.x+blockDim.x-1)/blockDim.x;
        }
        __syncthreads();

        __shared__ char haplotype_base_in_char[500];
        int hh=(read_haplotype_number.y+4-1)/4;
        int tt=(hh+blockDim.x-1)/blockDim.x;
        for(int ii=0;ii<tt;ii++)
        {
            int aa=threadIdx.x+ii*blockDim.x;
            if(aa< hh)
            {
                char4 haplotype_base_in_thread;
                haplotype_base_in_thread=haplotype_base_array[aa];
                haplotype_base_in_char[aa*4]=haplotype_base_in_thread.x;
                haplotype_base_in_char[aa*4+1]=haplotype_base_in_thread.y;
                haplotype_base_in_char[aa*4+2]=haplotype_base_in_thread.z;
                haplotype_base_in_char[aa*4+3]=haplotype_base_in_thread.w;
            }
        }
        __syncthreads();

        __shared__ float MM[130]; //left
        __shared__ float DD[130]; //left
        __shared__ float II[130]; //left
        __shared__ float MM2[130];
        __shared__ float DD2[130];
        __shared__ float II2[130];
        char read_base;
        float D_0=1.329228e+36/(float)read_haplotype_number.y;
        if(threadIdx.x==0)
        {
            MM[0]=0;
            DD[0]=D_0;
            II[0]=0;
            MM2[0]=0;
            II2[0]=0;
            DD2[0]=D_0;
        }
        int read_number=read_haplotype_number.x;
        int round_size;
        for(int i=0;i<round;i++)
        {
            round_size=(read_number>blockDim.x)?blockDim.x:read_number;
            read_number=(read_number>blockDim.x)?read_number-blockDim.x:0;
            char read_base;
            float M=1.0f;
            float Qm,Qm_1,alpha,beta,delta,epsion,xiksi;
            if(threadIdx.x<round_size ) // tid is from 0 ~ round_size-1
            {
                read_base=read_base_array[threadIdx.x+blockDim.x*i];
                delta=parameter_array[threadIdx.x+blockDim.x*i+skip];
                xiksi=parameter_array[threadIdx.x+blockDim.x*i+2*skip];
                alpha=parameter_array[threadIdx.x+blockDim.x*i+3*skip];
                epsion=0.1;
                beta=M-epsion;
                Qm=parameter_array[threadIdx.x+blockDim.x*i];
                Qm_1=M-Qm;
                Qm=fdividef(Qm,3.0f);
            }

            M=0;
            float I=0; //now
            float D=0; //now

            float MMID=0;
            float MMID2=0;
            float M2=0;
            float I2=0;
            float D2=0;

            int current_haplotype_id=0;
            for(int j=0;j<round_size+(read_haplotype_number.y+1)/2-1;j++)
            {
                int aa=j-threadIdx.x;
                if( aa>=0 && (current_haplotype_id<read_haplotype_number.y))
                {
                    if(i>0&&threadIdx.x==0)
                    {
                        MM[0]=MGG[current_haplotype_id];
                        II[0]=IGG[current_haplotype_id];
                        DD[0]=DGG[current_haplotype_id];
                        MM2[0]=MGG[current_haplotype_id+1];
                        II2[0]=IGG[current_haplotype_id+1];
                        DD2[0]=DGG[current_haplotype_id+1];
                    }

                    if(threadIdx.x==0 && i==0)
                        MMID2=__fmul_rn(beta,D_0);

                    float MID=__fadd_rn(II[threadIdx.x],DD[threadIdx.x]);
                    float DDM=__fmul_rn(M2,xiksi);
                    float IIMI=__fmul_rn(II[threadIdx.x],epsion);
                    float MIIDD=__fmul_rn(beta,MID);
                    char haplotype_base_each=haplotype_base_in_char[current_haplotype_id];
                    float aa=(haplotype_base_each==read_base)? Qm_1:Qm;

                    M=__fmul_rn(aa,MMID2);
                    D=__fmaf_rn(D2,epsion,DDM);
                    I=__fmaf_rn(MM[threadIdx.x],delta,IIMI);
                    MMID=__fmaf_rn(alpha,MM[threadIdx.x],MIIDD);

                    current_haplotype_id++;

                    if(threadIdx.x==round_size-1 && i==round-1)
                        result_block=__fadd_rn(result_block,__fadd_rn(M,I));

                    if(current_haplotype_id<read_haplotype_number.y)
                    {
                        MID=__fadd_rn(II2[threadIdx.x],DD2[threadIdx.x]);
                        DDM=__fmul_rn(M,xiksi);
                        IIMI=__fmul_rn(II2[threadIdx.x],epsion);

                        haplotype_base_each=haplotype_base_in_char[current_haplotype_id];
                        aa=(haplotype_base_each==read_base)? Qm_1:Qm;

                        MIIDD=__fmul_rn(beta,MID);
                        M2=__fmul_rn(aa,MMID);
                        I2=__fmaf_rn(MM2[threadIdx.x],delta, IIMI);
                        D2=__fmaf_rn(D,epsion,DDM);
                        MMID2=__fmaf_rn(alpha, MM2[threadIdx.x], MIIDD);

                        current_haplotype_id++;
                        if(threadIdx.x==round_size-1&& i==round-1)
                            result_block=__fadd_rn(result_block,__fadd_rn(M2,I2));
                    }

                }
                __syncthreads();

                MM[threadIdx.x+1]=M;
                DD[threadIdx.x+1]=D;
                II[threadIdx.x+1]=I;

                MM2[threadIdx.x+1]=M2;
                DD2[threadIdx.x+1]=D2;
                II2[threadIdx.x+1]=I2;
                if(i<round-1&&threadIdx.x==round_size-1 )
                {
                    MGG[current_haplotype_id-1]=M2;
                    IGG[current_haplotype_id-1]=I2;
                    DGG[current_haplotype_id-1]=D2;

                    MGG[current_haplotype_id-2]=M;
                    IGG[current_haplotype_id-2]=I;
                    DGG[current_haplotype_id-2]=D;
                }

                __syncthreads();
            }
        }


        if(threadIdx.x==round_size-1)
        {
            result[offset]=result_block;
        }
        offset+=gridDim.x;
    }
}

int runBatch(Batch& batch)
{

    float * MG;
    float * DG;
    float * IG;
    hipMalloc( (float **)& MG,sizeof(float) *240*500*3);
    DG=MG+240*500;
    IG=DG+240*500;

    int INI=(log10f((std::numeric_limits<float>::max() / 16)));

    int size;

    // ph2pr_h
    float ph2pr_h[BSIZE];
    for(int i=0;i<BSIZE;i++)
    {
        ph2pr_h[i]=powf(10.f, -((float)i) / 10.f);
    }


    while(!feof(file))
    {

        InputData *inputdata=(InputData* )malloc(size*(sizeof(InputData)));

        size = batch.num_reads * batch.num_haps;

        // align to BSIZE
        int mem_size=BALIGN(size*260); //read_base
        mem_size+=BALIGN(size*500); // haplotyp_base;
        mem_size+=BALIGN(size*260*4);//parameter1;
        mem_size+=BALIGN(size*260*4);//parameter2;
        mem_size+=BALIGN(size*260*4);//parameter3;
        mem_size+=BALIGN(size*260*4);//parameter4;
        mem_size+=BALIGN(size*260*4);//result;
        //mem_size+=(size* sizeof(float)+BSIZE-1)/BSIZE*BSIZE;//result;
        mem_size+=BALIGN(size*sizeof(NUM_ADD));//NUM_ADD;

        char *data_h_total=(char*)malloc(mem_size); //on cpu host

        // device
        char * result_d_total;
        hipError_t err;
        err=hipMalloc( (char **) &result_d_total, mem_size);
        if(err!=hipSuccess)
            printf( "Error %d: %s!\n", err, hipGetErrorString(err) );

        char * data_d_total=result_d_total+BALIGN(size*sizeof(float));  //on GPU

        char * data_h=data_h_total;
        char * data_h_begin=data_h;  //cpu
        NUM_ADD *data_num_add=(NUM_ADD *) (data_h); //cpu

        data_h=data_h+BALIGN(size*sizeof(NUM_ADD));

        int data_size=0;
        for(int i=0;i<size;i++)
        {
            int hi = i/batch.num_reads; // haplotype index
            int read_size=batch.reads[i].length;

            // parameter
            int skip=BALIGN(sizeof(float)*read_size)/sizeof(float);
            float parameter[1040];  // 260*4
            for(int j=0;j<read_size;j++)
            {
                parameter[j]= ph2pr_h[batch.reads[i].bases[j]&127 ];     //QM
                parameter[j+skip]=ph2pr_h[batch.reads[i].i[j]&127];      //Qi
                parameter[j+skip*2]=ph2pr_h[batch.reads[i].d[j]&127];    //QD
                parameter[j+skip*3]=1.0f-ph2pr_h[((int)(batch.reads[i].i[j]&127)+(int)(batch.reads[i].d[j]&127))&127];  // c
            }

            // haplotype
            int haplotype_new_size=(batch.haps[hi].length + 4-1)/4;
            char4 haplotype_base_new[150];
            for(int j=0;j<haplotype_new_size;j++)
            {
                haplotype_base_new[j].x = batch.haps[hi].bases[j*4];

                if(j*4+1 < batch.haps[hi].length)
                    haplotype_base_new[j].y = batch.haps[hi].bases[j*4+1];

                if(j*4+2 < batch.haps[hi].length)
                    haplotype_base_new[j].z = batch.haps[hi].bases[j*4+2];

                if(j*4+3 < batch.haps[hi].length)
                    haplotype_base_new[j].w = batch.haps[hi].bases[j*4+3];
            }

            // data_num_add on cpu
            data_num_add[i].read_haplotype_number.x=batch.reads[i].length;
            data_num_add[i].read_haplotype_number.y=batch.haps[hi].length; //inputdata[i].haplotype_size;
            data_num_add[i].address_array=data_size;

            char read_base_new[260];
            for(int j=0;j<read_size;j++)
            {
                read_base_new[j]=inputdata[i].read_base[j];
            }
            //read base
            memcpy(data_h,read_base_new,sizeof(char)*read_size);
            data_h+=(read_size+BSIZE-1)/BSIZE*BSIZE;
            data_size+=(read_size+BSIZE-1)/BSIZE*BSIZE;

            //parameter
            memcpy(data_h,parameter,sizeof(float) *skip*4);
            data_h+=sizeof(float) *skip*4;
            data_size+=sizeof(float) *skip*4;

            //haplotype
            memcpy(data_h,haplotype_base_new,sizeof(char4)* haplotype_new_size);
            data_h+=(haplotype_new_size*sizeof(char4)+BSIZE-1)/BSIZE*BSIZE;
            data_size+=(haplotype_new_size*sizeof(char4)+BSIZE-1)/BSIZE*BSIZE;
        }

        int data_size_to_copy=data_size+(size*sizeof(NUM_ADD)+BSIZE-1)/BSIZE*BSIZE;
        char * data_d;
        float * result_d=(float *) (result_d_total);
        int blocksize=BSIZE;
        int gridsize=180;
        NUM_ADD * num_add_d=(NUM_ADD *) (data_d_total);
        data_d=data_d_total+(sizeof(NUM_ADD)*size+BSIZE-1)/BSIZE*BSIZE;

        clock_gettime(CLOCK_MONOTONIC_RAW,&start);
        err=hipMemcpy(data_d_total,data_h_begin,data_size_to_copy,hipMemcpyHostToDevice);
        if(err!=hipSuccess)
            printf( "Error %d: %s!\n", err, hipGetErrorString(err) );
        pairHMM<<<gridsize,blocksize>>> (size,data_d,num_add_d,result_d,MG,DG,IG);
        //hipDeviceSynchronize();
        hipMemcpy (batch.results,result_d_total,size*sizeof(float),hipMemcpyDeviceToHost);
        clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
        computation_time+=diff(start,finish);

        free(data_h_total);
        err=hipFree(result_d_total);
        if(err!=hipSuccess)
            printf( "Error %d: %s!\n", err, hipGetErrorString(err) );

        for(int i=0;i<size;i++)
            float aa=(log10f((double)batch.results[i]) - INI);

        free(inputdata);
        free(result_h);
        fscanf(file,"%d",&size);
        //	if(total>10000)
//			break;
    }//end of while

    clock_gettime(CLOCK_MONOTONIC_RAW,&start);
    hipFree(MG);
    hipDeviceReset();
    clock_gettime(CLOCK_MONOTONIC_RAW,&finish);
    //mem_cpy_time+=diff(start,finish);//(finish1.tv_nsec-start1.tv_nsec)/1000000000.0;

    //	printf("size %d\n",total);
    printf("read_time=%e  initial_time=%e  computation_time= %e total_time=%e\n",read_time, data_prepare,computation_time, computation_time+mem_cpy_time+data_prepare);
    //	printf("%d %d %d  %e\n", fakesize, read_read, haplotype_haplotype,computation_time);
    printf("Total time=%e\n",total_time);
    return 0;
}

bool calculate(Batch& batch)
{

}